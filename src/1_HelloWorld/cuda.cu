#include "hip/hip_runtime.h"
#include<cudahelper.h>
#include<timer.h>
#include<vector3.cuh>
#include<vector>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include<stb_image_write.h>

using namespace std;

__global__ void render(color* fb, int maxX, int maxY) {
	int tidX = blockDim.x * blockIdx.x + threadIdx.x;
	int tidY = blockDim.y * blockIdx.y + threadIdx.y;

	if((tidX >= maxX) || (tidY >= maxY)) return;

	int pixel_index = tidY * maxX + tidX;
	fb[pixel_index] = color(float(tidX) / maxX, float(tidY) / maxY, 0.2f);
}

int main(int argc, char** argv) {
	const int imageWidth = 256;
	const int imageHeight = 256;

	timer t;
	t.start();

	color *fb;
	cudaerr(hipMallocManaged(&fb, imageWidth * imageHeight * 3 * sizeof(float)));
	int tx = 8, ty = 8;
	dim3 gridSize(imageWidth / tx + 1, imageHeight / ty + 1);
	dim3 blockSize(tx, ty);
	render<<<gridSize, blockSize>>>(fb, imageWidth, imageHeight);
	cudaerr(hipGetLastError());
	cudaerr(hipDeviceSynchronize());

	vector<unsigned char> pngData;
	for (int j = imageHeight - 1; j >= 0; j--) {
		for (int i = 0; i < imageWidth; i++) {
			size_t pixel_index = j * imageWidth + i;
			fb[pixel_index].addColor(pngData);
		}
	}
	t.end();
	stbi_write_png("outputcuda.png", imageWidth, imageHeight, 3, pngData.data(), imageWidth * 3);
	cout<<"Done. Time Taken = "<<t<<endl;

	hipFree(fb);
}